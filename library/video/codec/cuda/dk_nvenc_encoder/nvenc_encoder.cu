#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <memory.h>
#include "dk_cuda_driver_api.h"

/*
__device__ int device_clip_byte(unsigned char x)
{
	if (x > 255)
		return 255;
	else if (x < 0)
		return 0;
	else
		return x;
}

__global__ void kernel_convert_rgb32_to_yv12(unsigned int width, unsigned int height, unsigned char * rgba, unsigned int rgba_pitch,
											 unsigned char * y_plane, unsigned char * u_plane, unsigned char * v_plane, unsigned int y_pitch, unsigned int uv_pitch)
{
	int x, y;
	x = blockIdx.x*blockDim.x + threadIdx.x;
	y = blockIdx.y*blockDim.y + threadIdx.y;

	if ((x < width) && (y < height))
	{
		const unsigned char * pixel = rgba + rgba_pitch * x;
		y_plane[x] = device_clip_byte(((pixel[2] * 66 + pixel[1] * 129 + pixel[0] * 25 + 128) >> 8) + 16);
		if (y % 2 == 0 && x % 2 == 0) 
		{
			u_plane[x / 2] = device_clip_byte(((pixel[2] * -38 + pixel[1] * -74 + pixel[0] * 112 + 128) >> 8) + 128);
			v_plane[x / 2] = device_clip_byte(((pixel[2] * 112 + pixel[1] * -94 + pixel[0] * -18 + 128) >> 8) + 128);
		}
	}
}

void convert_rgb32_to_yv12(unsigned int width, unsigned int height, unsigned char * rgba, unsigned int rgba_pitch, 
						   hipDeviceptr_t y_plane, hipDeviceptr_t u_plane, hipDeviceptr_t v_plane, unsigned int y_pitch, unsigned int uv_pitch)
{
#define BLOCK_X 32
#define BLOCK_Y 16
	dim3 block(BLOCK_X, BLOCK_Y, 1); //512 block
	dim3 grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
#undef BLOCK_Y
#undef BLOCK_X




}
*/
__device__ unsigned char device_read_pixel_value(unsigned char * src, unsigned int width, int x, int y)
{
	return (unsigned char)src[y*width + x];
}

__device__ void device_put_pixel_value(unsigned char * dst, unsigned int width, int x, int y, unsigned char value)
{
	dst[y*width + x] = value;
}

__global__ void kernel_rotate( unsigned char * src, unsigned char * dst, int size_x, int size_y, float deg)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;// Kernel definition
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int xc = size_x - size_x / 2;
	int yc = size_y - size_y / 2;
	int newx = ((float)i - xc)*cos(deg) - ((float)j - yc)*sin(deg) + xc;
	int newy = ((float)i - xc)*sin(deg) + ((float)j - yc)*cos(deg) + yc;
	if (newx >= 0 && newx < size_x && newy >= 0 && newy < size_y)
	{
		device_put_pixel_value(dst, size_x, i, j, device_read_pixel_value(src, size_x, newx, newy));
	}
}

__global__ void kernel_interleave_uv(unsigned char *yuv_cb, unsigned char *yuv_cr, unsigned char *nv12_chroma, int chroma_width, int chroma_height, int cb_pitch, int cr_pitch, int nv12_pitch)
{
	int x, y;
	unsigned char *pCb;
	unsigned char *pCr;
	unsigned char *pDst;
	x = blockIdx.x*blockDim.x + threadIdx.x;
	y = blockIdx.y*blockDim.y + threadIdx.y;

	if ((x < chroma_width) && (y < chroma_height))
	{
		pCb = yuv_cb + (y*cb_pitch);
		pCr = yuv_cr + (y*cr_pitch);
		pDst = nv12_chroma + y*nv12_pitch;
		pDst[x << 1] = pCb[x];
		pDst[(x << 1) + 1] = pCr[x];
	}
}


#if defined(WITH_DYNAMIC_CUDA_LOAD)
void interleave_uv(void* driver_api, unsigned int width, unsigned int height, unsigned char * src, unsigned int src_pitch, hipDeviceptr_t dst, unsigned int dst_pitch, hipDeviceptr_t cb, hipDeviceptr_t cr)
#else
void interleave_uv(unsigned int width, unsigned int height, unsigned char * src, unsigned int src_pitch, hipDeviceptr_t dst, unsigned int dst_pitch, hipDeviceptr_t cb, hipDeviceptr_t cr)
#endif
{
	unsigned char * origin_yv12_y_plane = src;
	unsigned char * origin_yv12_v_plane = origin_yv12_y_plane + src_pitch*height;
	unsigned char * origin_yv12_u_plane = origin_yv12_v_plane + (src_pitch*height >> 2);

#if defined(WITH_DYNAMIC_CUDA_LOAD)
	dk_cuda_driver_api * driver_api_ = static_cast<dk_cuda_driver_api*>(driver_api);
#endif

	// copy luma
	hip_Memcpy2D copy_param;
	memset(&copy_param, 0, sizeof(copy_param));
	copy_param.dstMemoryType = hipMemoryTypeDevice;
	copy_param.dstDevice = dst;
	copy_param.dstPitch = dst_pitch;
	copy_param.srcMemoryType = hipMemoryTypeHost;
	copy_param.srcHost = origin_yv12_y_plane;
	copy_param.srcPitch = src_pitch;
	copy_param.WidthInBytes = width;
	copy_param.Height = height;
#if defined(WITH_DYNAMIC_CUDA_LOAD)
	driver_api_->memcpy_2d(&copy_param);
	driver_api_->memcpy_host_to_device(cb, origin_yv12_u_plane, src_pitch*height >> 2);
	driver_api_->memcpy_host_to_device(cr, origin_yv12_v_plane, src_pitch*height >> 2);

#else
	hipMemcpyParam2D(&copy_param);
	hipMemcpyHtoD(cb, origin_yv12_u_plane, src_pitch*height >> 2);
	hipMemcpyHtoD(cr, origin_yv12_v_plane, src_pitch*height >> 2);
#endif
	unsigned int chroma_height = height >> 1;
	unsigned int chroma_width = width >> 1;
#define BLOCK_X 32
#define BLOCK_Y 16
	dim3 block(BLOCK_X, BLOCK_Y, 1); //512 block
	dim3 grid((chroma_width + BLOCK_X - 1) / BLOCK_X, (chroma_height + BLOCK_Y - 1) / BLOCK_Y, 1);
#undef BLOCK_Y
#undef BLOCK_X

	unsigned int chroma_pitch = src_pitch >> 1;
	hipDeviceptr_t dst_chroma = (hipDeviceptr_t)((unsigned char*)dst + dst_pitch*height);
	kernel_interleave_uv << <block, grid >> >((unsigned char*)cb, (unsigned char*)cr, (unsigned char*)dst_chroma, chroma_width, chroma_height, chroma_pitch, chroma_pitch, dst_pitch);

	//hipError_t result = hipStreamQuery(NULL);
}