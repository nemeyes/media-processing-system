#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

__device__ __forceinline__ void rgb_to_y(const unsigned char b, const unsigned char g, const unsigned char uchar r, const unsigned char & y)
{
	y = static_cast<uchar>(((int)(30 * r) + (int)(59 * g) + (int)(11 * b)) / 100);
}

__device__ __forceinline__ void rgb_to_yuv(const unsigned char b, const unsigned char g, const unsigned char r, unsigned char & y, unsigned char & u, unsigned char & v)
{
	rgb_to_y(b, g, r, y);
	u = static_cast<uchar>(((int)(-17 * r) - (int)(33 * g) + (int)(50 * b) + 12800) / 100);
	v = static_cast<uchar>(((int)(50 * r) - (int)(42 * g) - (int)(8 * b) + 12800) / 100);
}

__global__ void convert_rgba_to_yv12_kernel(int width, int height)
{
	unsigned int x = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
	unsigned int y = (blockIdx.y * blockDim.y + threadIdx.y) * 2;

	if ((x + 1) >= width || (y + 1) > height)
		return;




	// load RGB values
	unsigned int R = input[(x + y * width) * RGBcomponentCount];
	unsigned int G = input[(x + y * width) * RGBcomponentCount + 1];
	unsigned int B = input[(x + y * width) * RGBcomponentCount + 2];

	// NTSC standard
	// wikipedia : http://en.wikipedia.org/wiki/YUV#Y.27UV420p_.28and_Y.27V12_or_YV12.29_to_RGB888_conversion
	float Y = 0.299 * R + 0.587 * G + 0.114 * B; 
	if (Y>255) Y = 255;
	float U = -0.147 * R - 0.289 * G + 0.436 * B + 128;
	if (U>255) U = 255;
	float V = 0.615 * R - 0.515 * G - 0.100 * B + 128;
	if (V>255) V = 255;
	output[x + y * width] = Y; // Y-components of all pixels
	output[(y / 2) * (width / 2) + (x / 2) + dst_size] = U; // U-components of all pixels
	output[(y / 2) * (width / 2) + (x / 2) + dst_size + (dst_size / 4)] = V; // V-components of all pixels
}

void cu_convert_rgba_to_yv12(int width, int height, unsigned char * rgba, unsigned char * yv12);
{
	unsigned char * cu_rgba;
	unsigned char * cu_yv12;

	hipMalloc((void**)&cu_rgba, 4 * width*height);
	hipMalloc((void**)&cu_yv12, 1.5*width*height);
	hipMemcpy(cu_rgba, rgba, 4 * width*height, hipMemcpyHostToDevice);



	hipMemcpy(yv12, cu_yv12, 1.5 * width*height, hipMemcpyDeviceToHost);
	hipFree(cu_rgb32);
	hipFree(cu_yv12);
}